#include "hip/hip_runtime.h"
// Rendering:
// 1) set the lens and precalculate constants - CPU and update to GPU
// 2) paint the screen white and reset the depth feild - Screen and Depth on GPU, done on GPU
// 3) for each triangle paint onto a pixel array - Done on GPU using GPU triangle data going to GPU 
// 4) copy that pixel array to the buffer frame
// 5) push the buffer array to the window
#include "gpuController.h"


void create_render_data(struct cpu_data **cpu_data, struct gpu_data **h_gpu_data, struct gpu_data **d_gpu_data, SDL_Surface *image, int nTrigs, const float *trigs, const Uint8 *clrs, float *v, float *o){
       // Holds all relevent dat about the objects, render constants, and imagescreen info
       //     Used to keep track of these things
       *cpu_data = (struct cpu_data *)malloc(sizeof(struct cpu_data));
       init_cpu_dat(*cpu_data, image, nTrigs, trigs, clrs, v, o);

       // Initalize relevent data onto the GPU -> stays for the duration of the program
       //                                      -> some parts can be eddited as we go
       //                                      -> Creates a memory space on the GPU
       // Allocate the pointer structure on the GPU and CPU -> list of pointers to GPU and other imediate data
       hipMalloc((void **)d_gpu_data, sizeof(struct gpu_data));
       *h_gpu_data = (struct gpu_data *)malloc(sizeof(struct gpu_data));
       init_gpu_dat(*d_gpu_data, *h_gpu_data, *cpu_data);
}

void init_cpu_dat(struct cpu_data *dat, SDL_Surface *image, int nTrigs, const float *trigs, const Uint8 *clrs, float *v, float *o){
       // Copy over the amount of trigs, and allocate space for personal use
       dat->nTrigs = nTrigs;
       dat->trigs = (float *)malloc(sizeof(float)*9*nTrigs);
       dat->colors = (Uint8 *)malloc(nTrigs*3);
       memcpy(dat->trigs, trigs, sizeof(float)*9*nTrigs);
       memcpy(dat->colors, clrs, 3*nTrigs);

       // Extract data about screen size and pixel organization, also dest pixel location
       dat->pixels_arr = (Uint8 *)image->pixels;
       dat->BPP = image->format->BytesPerPixel;
       dat->pitch = image->pitch;
       dat->w = image->w;
       dat->h = image->h;
       dat->pix_arr_size = (dat->h)*(dat->pitch);
       dat->depth_arr_size = (dat->h)*(dat->w)*sizeof(Uint32);

       // Allocate space for all equation constants -> keeping track of where we are and info to make projection easy
       dat->v = (float *)malloc(sizeof(float) * 15);
       dat->hx = dat->v + 3;
       dat->hy = dat->hx + 3;
       dat->c1 = dat->hy + 3;
       dat->mag = dat->c1 + 1;
       dat->offset = dat->mag + 1;
       dat->a = dat->offset + 3;


       // Update the equation constants based on the inital position
       update_lens(dat, v, o);
}

void update_lens(struct cpu_data *dat, float *v, float *o){
       cpyVec(v, dat->v);
       float temp[3] = {0,0,1};
       cpyVec(o, dat->offset);

       cross(dat->v, temp, dat->hx);
       normalize(dat->hx);
       //constMult(1, dat->hx, dat->hx)

       cross(dat->hx, dat->v, dat->hy);
       normalize(dat->hy);
       //constMult(1, dat->hy, dat->hy)

       *(dat->a) = 0.9;
       *(dat->mag) = vecMag(v);
       *(dat->mag) = (*(dat->mag)) * (*(dat->mag));
       *(dat->c1) = (1-*(dat->a)) * *(dat->mag);
       *(dat->c1) = *(dat->c1)/(2*vecMag(dat->hx));
}

void kill_cpu_data(struct cpu_data *dat){
       free(dat->trigs);
       free(dat->colors);
       free(dat->v);
       free(dat);
}

void init_gpu_dat(struct gpu_data *d_dat, struct gpu_data *h_dat, struct cpu_data *cpu_dat){
       
       // Fill all direct values
       struct gpu_data *temp = h_dat;
       temp->d_nTrigs = cpu_dat->nTrigs;
       temp->d_BPP = cpu_dat->BPP;
       temp->d_pitch = cpu_dat->pitch;
       temp->d_w = cpu_dat->w;
       temp->d_h = cpu_dat->h;

       // Allocate space for the pixel array
       int size = (cpu_dat->pitch)*(cpu_dat->h)*sizeof(Uint8);
       hipMalloc((void **)&(temp->d_pixels_arr), size);

       // Allocate space for the trig color data
       size = 3*cpu_dat->nTrigs;
       hipMalloc((void **)&(temp->d_colors), size);
       
       // Allocate space for depthscreen
       hipMalloc((void **)&(temp->d_depthScreen), temp->d_w*temp->d_h*sizeof(Uint32));

       // Allocate space for the rest of the floats
       //       = trigs and cords   constants
       size = (cpu_dat->nTrigs * 18 + 11 + 3)*sizeof(float);
       hipMalloc((void **)&(temp->d_trigs), size);

       // Set the rest of the pointers
       temp->d_cords_arr    = temp->d_trigs + cpu_dat->nTrigs * 9;
       temp->d_v            = temp->d_cords_arr + cpu_dat->nTrigs * 9;
       temp->d_hx           = temp->d_v + 3;
       temp->d_hy           = temp->d_hx + 3;
       temp->d_c1           = temp->d_hy + 3;
       temp->d_mag          = temp->d_c1 + 1;
       temp->d_offset       = temp->d_mag + 1;

       // copy over the triangles and the pointer list to the device
       hipMemcpy(temp->d_trigs, cpu_dat->trigs, sizeof(float) * cpu_dat->nTrigs * 9, hipMemcpyHostToDevice);
       hipMemcpy((void *)d_dat, (void *)temp, sizeof(struct gpu_data), hipMemcpyHostToDevice);
       hipMemcpy(temp->d_colors, cpu_dat->colors, 3*cpu_dat->nTrigs, hipMemcpyHostToDevice);
}

void kill_gpu_data(struct gpu_data *h_gDat){
       hipFree(h_gDat->d_pixels_arr);
       hipFree(h_gDat->d_depthScreen);
       hipFree(h_gDat->d_colors);
       hipFree(h_gDat->d_trigs);
       free(h_gDat);
}

void update_GPU_lens(struct gpu_data *h_dat, struct cpu_data *cdat){
       hipMemcpy(h_dat->d_v, cdat->v, 14*sizeof(float), hipMemcpyHostToDevice);
}

__device__ float dot(float *v, float *w, float *o){
       return (*v - *o)*(*w) + (*(v + 1) - *(o + 1))*(*(w + 1)) + (*(v + 2) - *(o + 2))*(*(w + 2));
}

__device__ void crossG(float *v, float *w, float *ans){
       ans[0] = v[1] * w[2] - w[1] * v[2];
       ans[1] = v[2] * w[0] - v[0] * w[2];
       ans[2] = v[0] * w[1] - w[0] * v[1];
}

__device__ void subG(float *v, float *w, float *ans){
       for(int i = 0; i < 3; i++){
              ans[i] = v[i] - w[i];
       }
}

__device__ float dist(float *w, float *v, float * o){
       float a = w[0] - o[0] - v[0];
       float b = w[1] - o[1] - v[1];
       float c = w[2] - o[2] - v[2];
       return sqrt(a*a + b*b + c*c);
}

__global__ void cordify(struct gpu_data *dat, int num_floats){ // Run for each vector j = float j*3 = i{
       
       // i is starting index of a vector
       int i = 3*(threadIdx.x + blockIdx.x * blockDim.x); //0, 3, 6

       float topx = *(float *)(dat->d_c1);
       float topy = topx;
       float bot =  *(float *)(dat->d_mag);
       float mag;

       

       // float *w = dat->d_trigs + i*3
       if(i < num_floats){
              // 1) calculate the x component

              topx = dot(dat->d_trigs + i, dat->d_hx, dat->d_offset) * topx;
              topy = dot(dat->d_trigs + i, dat->d_hy, dat->d_offset) * topy;
              bot = bot - dot(dat->d_trigs + i, dat->d_v, dat->d_offset);
              mag = dist(dat->d_trigs + i, dat->d_v, dat->d_offset);

              if(bot <= 0.1* *(float *)(dat->d_mag))
              { // behind
                     *(float *)(dat->d_cords_arr + i) = -1.0f;
                     *(float *)(dat->d_cords_arr + i + 1) = -1.0f;
                     *(float *)(dat->d_cords_arr + i + 2) = -1.0f;
              }
              else
              {

                     *(float *)(dat->d_cords_arr + i) = (topx/bot + 0.5)*dat->d_w;
                     *(float *)(dat->d_cords_arr + i + 1) = (topy/bot + 0.5)*dat->d_h;
                     *(float *)(dat->d_cords_arr + i + 2) = mag;
              }

       }
}

__device__ float max(float a, float b, float c, float d){
       return (((a > b) ? a:b) > ((c > d) ? c:d) ? ((a > b) ? a:b):((c > d) ? c:d));
}

__device__ float min(float a, float b, float c, float d){
       return (((a < b) ? a:b) < ((c < d) ? c:d) ? ((a < b) ? a:b):((c < d) ? c:d));
}


// Want to paint the area of the triangle given the cords in dat.
// First load in the triangle cordanates
// make a vector formula for each line joining the points
// P1P2, P2P3, P3P1
// Start at the minimium of y values -> to the maximum y value of the triangle
// for a given y0, find the range of x's you need to paint
// at x, y check the depth of the point and check if the points in range of the screen and then paint it

// Form is    t = a*y_0 + b
//            x = c*t   + d
// Side 0: P0->P1: v = P0 + t*(P1 - P0) --> (y_0 - P0_y)/(P1_y - P0_y) = t, x = P0_x + t*(P1_x - P0_x)
// Side 1: P1->P2:
 // Side 3: P2->P0
 
__global__ void draw(struct gpu_data *dat, int num_floats){
       // Run for each triangle

       int i = 9*(threadIdx.x + blockDim.x*blockIdx.x);



       if(i < num_floats){

              float points[3][3];
              float yMin, yMax, xMin, xMax;
              float as[3];
              float bs[3];
              float cs[3];
              float ds[3];
              float t;
              float xT;
              int Ix;
              int Iy;

              // Copy over the three points of the triangle into points from the coordanate data computed
              for(int j = 0; j < 3; j ++){
                     points[j][0] = dat->d_cords_arr[i + j*3];
                     points[j][1] = dat->d_cords_arr[i + j*3 + 1];
                     points[j][2] = dat->d_cords_arr[i + j*3 + 2];
              }
              if(points[0][2] != -1 && points[1][2] != -1 && points[2][2] != -1){

                     // Assign the begining and end of y values to traverse
                     yMin = min(points[0][1], points[1][1], points[2][1], 0);
                     yMax = max(points[0][1], points[1][1], points[2][1], dat->d_h);

                     // Determine the coeffcients to the Lines between the points to find x(y)
                     for(int j = 0; j < 3; j ++){
                            if(points[(j+1)%3][1] == points[j][1]){
                                   as[j] = 0;
                                   bs[j] = -1;
                            }
                            else{
                                   as[j] = 1/(points[(j+1)%3][1] - points[j][1]);
                                   bs[j] = -1 * points[j][1] * as[j];
                            }
                            cs[j] = points[(j+1)%3][0] - points[j][0];
                            ds[j] = points[j][0];
                     }
                     // What is the depth given the x, y cords -> will be in form az + bx + cy = d or a(z-z0) + b(x-x0) + x(y-y0) = 0
                     // Solve for z: z = d - bx - cy

                     // find depth array
                     float depth;
                     float c = points[0][1]*(points[2][0] - points[1][0]) - points[1][1]*(points[2][0] - points[0][0]) + points[2][1]*(points[1][0] - points[0][0]);
                     float b = points[2][0] - points[0][0];
                     float a;
                     if(c != 0 && b != 0){
                            c = (points[0][2]*(points[2][0] - points[1][0]) - points[1][2]*(points[2][0] - points[0][0]) + points[2][2]*(points[1][0] - points[0][0]))/c;
                            b = ((points[2][2] - points[0][2]) - c*(points[2][1] - points[0][1]))/b;
                            a = points[0][2] - b*points[0][0] - c*points[0][1];
                     }
                     else{
                            c = 0;
                            b = 0;
                            a = min(points[0][2], points[1][2], points[2][2], 10);// (points[0][2] + points[1][2] + points[2][2])/3;
                     }

                     // Traverse the pixels within the triangle
                     for(float y = yMin; y <= yMax; y+=0.9){
                            xMin =  99999999;
                            xMax = -99999999;
                            
                            // find the range of x values to use given the equations of the lines
                            for(int j = 0; j < 3; j++)
                            {
                                   t = as[j]*y + bs[j];
                                   xT = cs[j]*t + ds[j];
                                   if(t <= 1 && t >= 0)
                                   {
                                          if(xT < xMin){xMin = xT;}
                                          if(xT > xMax){xMax = xT;}
                                   }
                            }
                            xMin = (xMin < 0) ? 0: xMin;
                            xMax = (xMax > dat->d_w) ? dat->d_w:xMax;
                            
                            for(float x = xMin; x <= xMax; x+=0.9){
                                   Iy = dat->d_h - (int)y;
                                   Ix = (int)x;
                                   depth = (a + b*x + c*y)*1000;
                                   // Make sure that the pixel is within range of the screen and compare the depth.
                                   if(Iy >= 0 && Iy <= dat->d_h && Ix >= 0 && Ix <= dat->d_w){
                                          atomicMin(dat->d_depthScreen + Iy*dat->d_w + Ix, (int)depth);
                                          
                                          if(dat->d_depthScreen[Iy*dat->d_w + Ix] == (int)depth){
                                                 // Paint the pixel and set new depth
                                                 //dat->d_depthScreen[Iy*dat->d_w + Ix] = (int)depth;
                                                 dat->d_pixels_arr[Iy*dat->d_pitch + Ix*dat->d_BPP] = dat->d_colors[(int)(i/3)];
                                                 dat->d_pixels_arr[Iy*dat->d_pitch + Ix*dat->d_BPP + 1] = dat->d_colors[(int)(i/3) + 1];
                                                 dat->d_pixels_arr[Iy*dat->d_pitch + Ix*dat->d_BPP + 2] = dat->d_colors[(int)(i/3) + 2];

                                          }
                                   }
                            }
                            
                     }
              }

       }

}
void print_cords(int nTrigs, float *trigs){
       printf("\nPrinting object data for %d triangles...\n", nTrigs);
       for(int i = 0; i < nTrigs; i++){
              printf("Triangle %d:\n\t P1: (%f, %f, %f) \n\t P2: (%f, %f, %f) \n\t P3:(%f, %f, %f) \n\n",
                     i, trigs[9*i], trigs[9*i + 1], trigs[9*i + 2], trigs[9*i + 3],
                     trigs[9*i + 4], trigs[9*i + 5], trigs[9*i + 6], trigs[9*i + 7],
                     trigs[9*i + 8]);
       }
}
void render_and_buffer(struct gpu_data *d_gDat, struct gpu_data *h_gDat, struct cpu_data *cDat){
       
       // 1. Paint pixels white and reset depth array to max
       hipMemset(h_gDat->d_pixels_arr, (Uint8)(0), cDat->pix_arr_size);
       hipMemset(h_gDat->d_depthScreen, (Uint8)(0xFF), cDat->depth_arr_size);

       // Turn all triangle vectors into cords
       int n = cDat->nTrigs;
       dim3 grid_size( (int)(n*3 / 512) + 1); // 5000
       dim3 block_size(512);

       cordify<<<grid_size, block_size>>>(d_gDat, n*9);//<<<1,1>>>(h_gDat->d_cords_arr);
       
       float *temp = (float *)malloc(sizeof(float)*n*9);
       hipMemcpy(temp, h_gDat->d_cords_arr, sizeof(float)*9*n, hipMemcpyDeviceToHost);
       //print_cords(n, temp);
       free(temp);

       // draw each triangle
       dim3 gridsize2((int)(n/512) + 1);
       draw<<<gridsize2, 512>>>(d_gDat, n*9);

       // Copy the pixel array back to the window
       hipMemcpy(cDat->pixels_arr, h_gDat->d_pixels_arr, (cDat->pix_arr_size)*sizeof(Uint8), hipMemcpyDeviceToHost);


}



