
#include "renderer_cuda.h"


void passert_cuda(bool cond, static char *msg){
       if(!cond){
              printf("CUDA ERROR: %s: %s\n", msg, hipGetErrorString(hipGetLastError()));
              exit(1);
       }
}


void safe_cudaAlloc(void **dest, uint32_t size, static char *msg){
       hipMalloc(dest, size);
       if(*dest == NULL){
              printf("CUDA ALLOCATION ERROR: %s: %s", msg, hipGetErrorString(hipGetLastError()));
              exit(1);
       }
}


typedef struct asset_t{    // array struct
       uint32_t      nTrigs;
       float         *trigs;
       uint8_t       *colors;
} asset_t;



typedef struct asset_record_t {    // Linked list struct
       uint32_t             nTrigs;
       float                *d_trigs;
       uint8_t              *d_colors;
       asset_record_t       *next;
} asset_record_t;



typedef struct static_render_data_t{
       // output frame buffer + info
       uint8_t       *pixels_arr;
       uint32_t      *depthScreen_arr;
       uint8_t       BPP;
       uint32_t      pitch;
       uint32_t      w;
       uint32_t      h;

       // Stored assets
       asset_t      *asset_pointer_arr;                           // Variable array -> needs to be resized on addtions

} static_render_data_t;


typedef struct instance2_t {
       int    asset_id;
       float  *buffer_loc;

       float offset[3];
} instance2_t;


typedef struct dynamic_render_data2_t{
       float offset[3];
       float view[3];

       instance2_t *instances_arr;
};




typedef struct gpu_allocations_t{

       static_render_data_t *d_static_data;
       static_render_data_t static_data_local_copy;
       uint32_t             *d_depthScreen_arr;
       uint8_t              *d_pixels_arr;

       int                   asset_arr_size;
       int                   n_assets;
       asset_record_t       *asset_record_head;         // linked list of other allocations
       asset_t              *d_asset_pointer_arr;


       dynamic_render_data2_t      *d_dynamic_data;

       uint32_t                    n_instances;
       uint32_t                    sz_buffer;
       instance2_t                 *instance_pointer_arr;
       float                       *d_cord_arr_buffer;

} gpu_allocations_t;





static gpu_allocations_t *allocs;


void add_record(asset_record_t *head, asset_record_t *new_record){
       if(head == NULL){
              return ;
       }
       while(head->next != NULL){
              head = head->next;
       }
       head->next = new_record;
}

asset_record_t *get_nth_record(asset_record_t *head, int n){
       while(head != NULL && n != 0){
              n = n - 1;
              head = head->next;
       }
       return head;
}



int init(SDL_Surface *image){
       allocs = (gpu_allocations_t *)calloc(sizeof(gpu_allocations_t), 1);
       passert(allocs != NULL, "Initing cuda renderer");

       static_render_data_t srd;

       srd.pixels_arr       = NULL;
       srd.depthScreen_arr  = NULL;
       srd.BPP              = image->format->BytesPerPixel;
       srd.pitch            = image->pitch;
       srd.w                = image->w;
       srd.h                = image->h;
       srd.asset_pointer_arr= NULL;

       int pixel_arr_size = srd.pitch*srd.h*sizeof(Uint8);
       int depth_arr_size = srd.w*srd.h*sizeof(uint32_t);

       safe_cudaAlloc((void **)&(allocs->d_depthScreen_arr), depth_arr_size, "Depth array");

       safe_cudaAlloc((void **)&(allocs->d_pixels_arr), pixel_arr_size, "Pixel array");

       safe_cudaAlloc((void **)&(allocs->d_static_data), sizeof(static_render_data_t), "Static render data");

       safe_cudaAlloc((void **)&(allocs->d_dynamic_data), sizeof(dynamic_render_data2_t), "Dynamic render data");
       
       srd.depthScreen_arr = allocs->d_depthScreen_arr;
       srd.pixels_arr = allocs->d_pixels_arr;

       memcpy(&(allocs->static_data_local_copy), &srd, sizeof(static_render_data_t));
       hipMemcpy(allocs->d_static_data, &srd, sizeof(static_render_data_t), hipMemcpyHostToDevice);

       return 0;
}

int alloc_asset(float *trigs, uint8_t *colors, uint32_t nTrigs){

       // 1) Allocate space for new asset (a) space in array for pointer - maybe, (b) trig and color arr - always
       if(allocs->asset_arr_size == allocs->n_assets){
              // Need to resize array
              asset_t *new_allocation;
              int old_arr_size = allocs->n_assets;
              int new_arr_size = allocs->n_assets * 2 + 1;

              safe_cudaAlloc((void **)&(new_allocation), sizeof(asset_t)*new_arr_size, "Resizing asset array");
              hipMemcpy(new_allocation, allocs->d_asset_pointer_arr, old_arr_size, hipMemcpyDeviceToDevice); // Making own kernel would be faster apperently 
              hipFree(allocs->d_asset_pointer_arr);
              allocs->d_asset_pointer_arr = new_allocation;
              
              // Copy over the new sigular pointer to the asset array
              allocs->static_data_local_copy.asset_pointer_arr = new_allocation;
              hipMemcpy(allocs->d_static_data, &(allocs->static_data_local_copy), sizeof(static_render_data_t), hipMemcpyHostToDevice);

       }
       allocs->n_assets = allocs->n_assets + 1;


       // Either way will need to allocate new gpu space for trigs and colors
       int trig_arr_size = nTrigs * sizeof(float) * 9;
       int color_arr_size = nTrigs * sizeof(uint8_t) * 3;

       // keep the record of the allocation
       asset_record_t *new_asset_record = (asset_record_t *)calloc(sizeof(asset_record_t), 1);
       new_asset_record->nTrigs = nTrigs;
       new_asset_record->next = NULL;
       if(allocs->asset_record_head == NULL){
              allocs->asset_record_head = new_asset_record;
       } else{
              add_record(allocs->asset_record_head, new_asset_record);
       
       }
       safe_cudaAlloc((void **)&(new_asset_record->d_trigs), trig_arr_size, "New asset trianlge array");
       safe_cudaAlloc((void **)&(new_asset_record->d_colors), color_arr_size, "New asset color array");


       asset_t gpu_asset;
       gpu_asset.nTrigs = nTrigs;
       gpu_asset.trigs = new_asset_record->d_trigs;
       gpu_asset.colors = new_asset_record->d_colors;


       // 2) copy over the data 
       hipMemcpy(gpu_asset.trigs, trigs, trig_arr_size, hipMemcpyHostToDevice);
       hipMemcpy(gpu_asset.colors, colors, color_arr_size, hipMemcpyHostToDevice);
       hipMemcpy(allocs->d_asset_pointer_arr + allocs->n_assets - 1, &gpu_asset, sizeof(asset_t), hipMemcpyHostToDevice);
       

       return 0;
}


int update_dynamic_data(float *view, float *offset, instance_t *inst_data, int n_instances){
       //hipMemcpy(allocs->d_dynamic_data, dyn_rend_data, sizeof(dynamic_render_data2_t), hipMemcpyHostToDevice);
       
       dynamic_render_data2_t new_dynamic_rd = {0};
       cpyVec(view, new_dynamic_rd.view);
       cpyVec(offset, new_dynamic_rd.offset);


       
       instance_t *head = inst_data;
       if(head == NULL){
              hipMemcpy(allocs->d_dynamic_data, &new_dynamic_rd, sizeof(dynamic_render_data2_t), hipMemcpyHostToDevice);
              return 0;
       }

       // Get the size of buffer needed for this amount of instances
       asset_record_t *asst_head = allocs->asset_record_head;
       asset_record_t *asset;
       uint32_t inst_sum;
       int i = 0;
       while(head != NULL){
              asset = get_nth_record(asst_head, head->asset_id);
              if(asset == NULL){
                     return -1;
              }
              inst_sum = asset->nTrigs*9;
              head = head->next;
              i += 1;
       }
       if(n_instances != i){
              return -1;
       }

       // Make a bigger buffer if needed
       if(allocs->sz_buffer < inst_sum){
              float *new_allocation_cord = NULL;
              safe_cudaAlloc((void **)&(new_allocation_cord), inst_sum*sizeof(float), "Resizing cord/buffer array");
              hipFree(allocs->d_cord_arr_buffer);
              allocs->d_cord_arr_buffer = new_allocation_cord;
       }

       // Make a bigger buffer if needed
       if(allocs->n_instances < n_instances){
              instance2_t *new_allocation_inst = NULL;
              safe_cudaAlloc((void **)&(new_allocation_inst), n_instances*sizeof(instance2_t), "Resizing instnace array");
              hipFree(allocs->instance_pointer_arr);
              allocs->instance_pointer_arr = new_allocation_inst;
       }

       // Now allocate buffer space for each instance
       new_dynamic_rd.instances_arr = allocs->instance_pointer_arr;
       
       instance2_t new_inst;
       uint32_t running_offset = 0;
       head = inst_data;
       for(int i = 0; i < n_instances; i += 1){
              new_inst.asset_id = head->asset_id;
              cpyVec(head->offset, new_inst.offset);
              new_inst.buffer_loc = allocs->d_cord_arr_buffer + running_offset;

              asset = get_nth_record(asst_head, head->asset_id);
              running_offset = running_offset + asset->nTrigs*9;
              head = head->next;
              hipMemcpy(allocs->instance_pointer_arr + i, &new_inst, sizeof(instance2_t), hipMemcpyHostToDevice);

       }

       hipMemcpy(allocs->d_dynamic_data, &new_dynamic_rd, sizeof(dynamic_render_data2_t), hipMemcpyHostToDevice);

       return 0;
}



void *get_device_trig_pointer(){
       if(allocs == NULL){
              return NULL;
       }
       passert(allocs->asset_record_head != NULL, "There are no asset records");
       return (void *)allocs->asset_record_head->d_trigs;
}


/*
[static data]
       [depth], [pixel], [assetp, assetp, assetp]

[dynamic data]



*/

int kill(){
       hipFree(allocs->d_static_data);
       hipFree(allocs->d_depthScreen_arr);
       hipFree(allocs->d_pixels_arr);
       hipFree(allocs->d_asset_pointer_arr);
       hipFree(allocs->d_dynamic_data);


       asset_record_t *cur_asset_rec = allocs->asset_record_head;
       asset_record_t *prev_asset_rec;

       while(cur_asset_rec != NULL){
              hipFree(cur_asset_rec->d_trigs);    // Free the trig array
              hipFree(cur_asset_rec->d_colors);   // Free color array
              prev_asset_rec = cur_asset_rec;
              cur_asset_rec = cur_asset_rec->next;
              free(prev_asset_rec);              // Free this record
       }
       free(allocs);
       return 0;
}